#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hip/hip_math_constants.h>

#include "pushRelabelGPU.h"

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
        bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

#define IDX(i, j, n) ((i) * (n) + (j))

#define UPDIV(n, d)   (((n) + (d) - 1) / (d))

static dim3 threadsPerBlock(1024, 1, 1);

__global__ void pushRelabelLockFreeKernel(int *residualFlow,
        int *height, int *excessFlow, int *netFlowOutS, int *netFlowInT, 
        int s, int t, int n) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    int u = index;
    if (u >= s) {
        u++;
    }
    if (u >= t) {
        u++;
    }

    // one thread here for all vertices not s or t
    while (*netFlowOutS != *netFlowInT) {
        if (u < n && excessFlow[u] > 0) {
            int curExcess = excessFlow[u];
            int curLowestNeighbor = -1;
            int neighborMinHeight = (int) HIP_INF;
            for (int v = 0; v < n; v++) {
                if (u == v) continue;
                if (residualFlow[IDX(u, v, n)] > 0) {
                    int tempHeight = height[v];
                    if (tempHeight < neighborMinHeight) {
                        curLowestNeighbor = v;
                        neighborMinHeight = tempHeight;
                    }
                }
            }
            if (height[u] > neighborMinHeight) {
                int delta = min(curExcess, residualFlow[IDX(u, curLowestNeighbor, n)]);
                atomicSub(&residualFlow[IDX(u, curLowestNeighbor, n)], delta);
                atomicAdd(&residualFlow[IDX(curLowestNeighbor, u, n)], delta);
                atomicSub(&excessFlow[u], delta);
                atomicAdd(&excessFlow[curLowestNeighbor], delta);
                if (curLowestNeighbor == s) {
                    atomicSub(netFlowOutS, delta);
                } else if (curLowestNeighbor == t) {
                    atomicAdd(netFlowInT, delta);
                }
            } else {
                height[u] = neighborMinHeight + 1;
            }
        }
    }
}

// Push-relabel algorithm to find max s-t flow. Based on lock-free implementation
// specified by Bo Hong. Uses one CUDA thread per vertex.
Flow *pushRelabelLockFreeGPU(Graph *g, int s, int t) {
    int *residualFlow;
    int *height;
    int *excessFlow;
    int *netFlowOutS;
    int *netFlowInT;
    int *tempHeights = (int *)calloc(g->n,  sizeof(int));
    int *tempExcessFlows = (int *)calloc(g->n,  sizeof(int));
    int *finalFlow = (int *)malloc((g->n * g->n) * sizeof(int));
    memcpy(finalFlow, g->capacities, (g->n * g->n) * sizeof(int));

    cudaCheckError(hipMalloc((void **)&residualFlow, sizeof(int) * (g->n * g->n)));
    cudaCheckError(hipMalloc((void **)&height, sizeof(int) * g->n));
    cudaCheckError(hipMalloc((void **)&excessFlow, sizeof(int) *  g->n));
    cudaCheckError(hipMalloc((void **)&netFlowOutS, sizeof(int)));
    cudaCheckError(hipMalloc((void **)&netFlowInT, sizeof(int)));

    // initialize preflow
    int flowOutS = 0;
    int flowInT = 0;
    tempHeights[s] = g->n;
    #pragma omp parallel for reduction(+:flowOutS)
    for (int v = 0; v < g->n; v++) {
        int cap = g->capacities[IDX(s, v, g->n)];
        if (cap > 0 && (s != v)) {
            finalFlow[IDX(s, v, g->n)] = 0;
            finalFlow[IDX(v, s, g->n)] += cap;
            flowOutS += cap;
            tempExcessFlows[v] = cap;
            if (v == t) {
                flowInT += cap;
            }
        }
    }

    cudaCheckError(hipMemcpy(residualFlow, finalFlow, sizeof(int) * (g->n * g->n),
                   hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(height, tempHeights, sizeof(int) * g->n,
                   hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(excessFlow, tempExcessFlows, sizeof(int) * g->n,
                   hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(netFlowInT, &flowInT, sizeof(int),
                   hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(netFlowOutS, &flowOutS, sizeof(int),
                   hipMemcpyHostToDevice));

    int numBlocks = UPDIV((g->n - 2), threadsPerBlock.x);
    pushRelabelLockFreeKernel<<<numBlocks, threadsPerBlock>>>(residualFlow,
        height, excessFlow, netFlowOutS, netFlowInT, s, t, g->n);

    free(tempHeights);
    free(tempExcessFlows);

    cudaCheckError(hipDeviceSynchronize());

    cudaCheckError(hipMemcpy(finalFlow, residualFlow, sizeof(int) * (g->n * g->n),
               hipMemcpyDeviceToHost));

    cudaCheckError(hipMemcpy(&flowInT, netFlowInT, sizeof(int),
               hipMemcpyDeviceToHost));

    // now update flow to represent actual flow
    #pragma omp parallel for schedule(static)
    for (int i = 0; i < (g->n * g-> n); i++) {
        finalFlow[i] = g->capacities[i] - finalFlow[i];
    }

    Flow *result = (Flow *)malloc(sizeof(Flow));
    result->maxFlow = flowInT;
    result->finalEdgeFlows = finalFlow;
    cudaCheckError(hipFree(residualFlow));
    cudaCheckError(hipFree(height));
    cudaCheckError(hipFree(excessFlow));
    cudaCheckError(hipFree(netFlowOutS));
    cudaCheckError(hipFree(netFlowInT));
    return result;
}